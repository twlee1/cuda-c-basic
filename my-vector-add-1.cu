
#include <hip/hip_runtime.h>
#include <stdio.h>

//__global__ void mykernel(void) {i
__global__ void add(int *a, int *b, int *c){
	*c = *a + *b;	
}

int main (void) {

	int a, b, c;
	int *d_a, *d_b, *d_c;
	int size=sizeof(int);

	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	a=2;
	b=7;
	
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	add<<<1,1>>>(d_a, d_b, d_c);

	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	printf("c=%d\n", c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);


	//mykernel<<<1,1>>>();

	//printf("Hello world\n");
	return 0;
}
