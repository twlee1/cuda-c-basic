
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 512

__global__ void add(int *a, int *b, int *c){
	c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];	
}

void random_ints(int* x, int size)
{
	for (int i=0; i<size; i++){
		x[i]=rand()%10;
	}
}

int main (void) {

	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size= N * sizeof(int);

	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	a = (int*)malloc(size); 
	random_ints(a, N);
	b = (int*)malloc(size); 
	random_ints(b, N);
	c = (int*)malloc(size); 

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	add<<<1,N>>>(d_a, d_b, d_c);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	for (int i=0; i<N; i++){
		printf("[%d] a:%d + b:%d = c:%d\n", i, a[i], b[i], c[i]);
	}

	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);

	return 0;
}
