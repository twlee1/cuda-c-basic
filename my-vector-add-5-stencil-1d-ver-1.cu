
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define RADIUS 3
#define BLK_SIZE 256
#define NUM_ELEMENTS (BLK_SIZE * 32)	// 256 * 32 = 8192


__global__ void stencil_1d(int *d_in, int *d_out){
	int gindex = (blockIdx.x * blockDim.x) + threadIdx.x + RADIUS;

	int result = 0;
	for (int offset = -RADIUS; offset <= RADIUS; offset++)
		result += d_in[gindex + offset];

	d_out[gindex - RADIUS] = result;
}


int main(void){
	int h_in[ NUM_ELEMENTS + (2*RADIUS) ];
	int h_out[ NUM_ELEMENTS ];	
	int *d_in, *d_out;

	// Initialize host input values
	for (int i=0; i<(NUM_ELEMENTS + 2*RADIUS); i++)
		h_in[i] = 1;

	// Allocate device global memory
	hipMalloc( &d_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int) );
	hipMalloc( &d_out, NUM_ELEMENTS * sizeof(int) );

	// Copy HOST -> DEVICE
	hipMemcpy( d_in, h_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int), hipMemcpyHostToDevice);

	// Launch kernel
	stencil_1d<<< NUM_ELEMENTS/BLK_SIZE, BLK_SIZE>>>(d_in, d_out);
	// NUM_ELEMENTS / BLK_SIZE = 8192 / 256 = 32
	// BLK_SIZE = 256

	// Copy result DEVICE -> HOST
	hipMemcpy( h_out, d_out, NUM_ELEMENTS * sizeof(int), hipMemcpyDeviceToHost);

	// Verify results
	int err_cnt = 0;
	for (int i=0; i<NUM_ELEMENTS; i++){
		if (h_out[i] != 7){
			printf("h_out[%d] == %d != 7\n", i, h_out[i]);
			err_cnt++;
			break;
		}
	}
	if (err_cnt!=0){ 
		printf("Wrong result\n"); 
	}else{
		printf("Success\n");
	}

	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
